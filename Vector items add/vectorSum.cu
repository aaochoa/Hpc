//============================================================================
#include<cstdlib>
#include<time.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include<math.h>
#define BLOCK_SIZE 1024 // Because it's just an array, 1 dimension

using namespace std;
//====== Serial vector ADD =====================================================
double serialVectorItemsAdd (double *A, int length)
{
  double sum=0;

  for (int i = 0; i < length; i++)
  {
    sum = sum + A[i];
  }
  return sum;
}

//====== Function made to print vector =========================================
void printVector (double *A, int length)
{
  for (int i=0; i<length; i++)
  {
    cout<<A[i]<<" | ";
  }
  cout<<endl;
}

//====== Function made to fill the vector with some given value ================
void fillVector (double *A, double value, int length)
{
  for (int i=0; i<length; i++)
  {
    A[i] = value;
  }
}

//====== To compare both results parallel and serial ===========================
void resultCompare(double A, double  *B)
{
  if(fabs(A-B[0]) < 0.1)
  {
    cout<<"Well Done"<<endl;
  } else
  {
    cout<<"Not working"<<endl;
  }
}

//======= Reduction kernel =====================================================
//Parallel
__global__ void reduceKernel(double *g_idata, double *g_odata, int length)
{
  __shared__ double sdata[BLOCK_SIZE];
  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<length)
  {
    sdata[tid] = g_idata[i];
  } else
  {
    sdata[tid] = 0.0;
  }
  __syncthreads();
  // do reduction in shared mem
  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0)
  {
    g_odata[blockIdx.x] = sdata[0];

  }
}
//====== Function made to call the reduction kernel ============================
void vectorItemsAdd(double *A, double *B, int length)
{
  double * d_A;
  double * d_B;
  double * algo = (double *) malloc(length * sizeof(double));

  hipMalloc(&d_A,length*sizeof(double));
  hipMalloc(&d_B,length*sizeof(double));

  hipMemcpy(d_A, A,length*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(d_B, B,length*sizeof(double),hipMemcpyHostToDevice);

  int aux=length;
  while(aux>1)
  {
     dim3 dimBlock(BLOCK_SIZE,1,1);
     int grid=ceil(aux/(double)BLOCK_SIZE);
      dim3 dimGrid(grid,1,1);
     reduceKernel<<<dimGrid,dimBlock>>>(d_A,d_B,aux);
     hipDeviceSynchronize();
     hipMemcpy(d_A,d_B,length*sizeof(double),hipMemcpyDeviceToDevice);
     aux=ceil(aux/(double)BLOCK_SIZE);
  }

  hipMemcpy(B,d_B,length*sizeof(double),hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
}

//======= MAIN function ========================================================
int main ()
{
 for(int i=0; i<=29;i++)//to execute the program many times
 {	cout<<"=> EXECUTION #"<<i<<endl;
  	unsigned int l = pow(2,i); //Vector's length
  	cout<<"Matrix size= "<<l<<endl;
 		clock_t start, finish;
 		double elapsedSecuential, elapsedParallel, optimization;
   	double *A = (double *) malloc(l * sizeof(double));
   	double *B = (double *) malloc(l * sizeof(double));

   fillVector(A,1.0,l);
   fillVector(B,0.0,l);

   start = clock();
   double sum = serialVectorItemsAdd(A,l);
   finish = clock();
   cout<< "The result is: " << sum << endl;
   elapsedSecuential = (((double) (finish - start)) / CLOCKS_PER_SEC );
   cout<< "The Secuential process took: " << elapsedSecuential << " seconds to execute "<< endl<< endl;

   start = clock();
   vectorItemsAdd(A,B,l);
   finish = clock();
   cout<< "The result is: " << B[0] << endl;
   elapsedParallel = (((double) (finish - start)) / CLOCKS_PER_SEC );
   cout<< "The Parallel process took: " << elapsedParallel << " seconds to execute "<< endl<< endl;

   optimization = elapsedSecuential/elapsedParallel;
   cout<< "The acceleration we've got: " << optimization <<endl;

   resultCompare(sum, B);
	 cout<< "============================================ "<<endl;

   free(A);
   free(B);
 }
}
