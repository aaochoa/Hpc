#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<cstdlib>
#include<hip/hip_runtime.h>
#include<highgui.h>
#include<cv.h>

#define N_elements 32
#define Mask_size  3
#define TILE_SIZE  1024
#define BLOCK_SIZE 32
__constant__ char M[Mask_size*Mask_size];
using namespace std;
using namespace cv;


__device__ unsigned char delimit(int value)//__device__ because it's called by a kernel
{
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return  value;
}



__global__ void convolution2DGlobalMemKernel(unsigned char *In,char *M, unsigned char *Out,int Mask_Width,int Rowimg,int Colimg)
{

   unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

   int Pvalue = 0;

   int N_start_point_row = row - (Mask_Width/2);
   int N_start_point_col = col - (Mask_Width/2);

   for(int i = 0; i < Mask_Width; i++)
   {
       for(int j = 0; j < Mask_Width; j++ )
       {
        if((N_start_point_col + j >=0 && N_start_point_col + j < Rowimg)&&(N_start_point_row + i >=0 && N_start_point_row + i < Colimg))
        {
          Pvalue += In[(N_start_point_row + i)*Rowimg+(N_start_point_col + j)] * M[i*Mask_Width+j];
        }
       }
   }

   Out[row*Rowimg+col] = delimit(Pvalue);

}

__global__ void convolution2DConstantMemKernel(unsigned char *In,unsigned char *Out,int Mask_Width,int Rowimg,int Colimg)
 {
   unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

   int Pvalue = 0;

   int N_start_point_row = row - (Mask_Width/2);
   int N_start_point_col = col - (Mask_Width/2);

   for(int i = 0; i < Mask_Width; i++)
   {
       for(int j = 0; j < Mask_Width; j++ )
       {
         if((N_start_point_col + j >=0 && N_start_point_col + j < Rowimg)&&(N_start_point_row + i >=0 && N_start_point_row + i < Colimg))
         {
           Pvalue += In[(N_start_point_row + i)*Rowimg+(N_start_point_col + j)] * M[i*Mask_Width+j];
         }
       }
    }

   Out[row*Rowimg+col] = delimit(Pvalue);
}

void convolution2DGlobalMemKernelCall(Mat image,unsigned char *In,unsigned char *Out,char *h_Mask,int Mask_Width,int Row,int Col){
  // Variables
  int Size_of_bytes =  sizeof(unsigned char)*Row*Col*image.channels();
  int Mask_size_bytes =  sizeof(char)*9;
  unsigned char *d_In, *d_Out;
  char *d_Mask;
  float Blocksize=BLOCK_SIZE;


  // Memory Allocation in device
  hipMalloc((void**)&d_In,Size_of_bytes);
  hipMalloc((void**)&d_Out,Size_of_bytes);
  hipMalloc((void**)&d_Mask,Mask_size_bytes);
  // Memcpy Host to device
  hipMemcpy(d_In,In,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Out,Out,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Mask,h_Mask,Mask_size_bytes,hipMemcpyHostToDevice);

  dim3 dimGrid(ceil(Row/Blocksize),ceil(Col/Blocksize),1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  convolution2DGlobalMemKernel<<<dimGrid,dimBlock>>>(d_In,d_Mask,d_Out,Mask_Width,Row,Col);

  hipDeviceSynchronize();
  // save output result.
  hipMemcpy (Out,d_Out,Size_of_bytes,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
  hipFree(d_Mask);
}

void convolution2DConstantMemKernelCall(Mat image,unsigned char *In,unsigned char *Out,char *h_Mask,int Mask_Width,int Row,int Col){
  // Variables
  int Size_of_bytes =  sizeof(unsigned char)*Row*Col*image.channels();
  int Mask_size_bytes =  sizeof(char)*9;
  unsigned char *d_In, *d_Out;
  char *d_Mask;
  float Blocksize=BLOCK_SIZE;


  // Memory Allocation in device
  hipMalloc((void**)&d_In,Size_of_bytes);
  hipMalloc((void**)&d_Out,Size_of_bytes);
  hipMalloc((void**)&d_Mask,Mask_size_bytes);
  // Memcpy Host to device
  hipMemcpy(d_In,In,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Out,Out,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M),h_Mask,Mask_size_bytes);
  dim3 dimGrid(ceil(Row/Blocksize),ceil(Col/Blocksize),1);
  dim3 dimBlock(Blocksize,Blocksize,1);
  convolution2DConstantMemKernel<<<dimGrid,dimBlock>>>(d_In,d_Out,Mask_Width,Row,Col);

  hipDeviceSynchronize();
  // save output result.
  hipMemcpy (Out,d_Out,Size_of_bytes,hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_In);
  hipFree(d_Out);
}

int main()
{

  clock_t start, finish; //Clock variables
  double elapsedParallel,elapsedParallelConstant;
  int Mask_Width =  Mask_size;
  int op = 2; // To select which parallel version we want to execute
  Mat image;
  image = imread("inputs/img1.jpg",0);   // Read the file
  Size s = image.size();
  int Row = s.width;
  int Col = s.height;
  char h_Mask[] = {-1,-1,-1,0,0,0,1,1,1}; // A kernel for edge detection
  unsigned char *img = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());
  unsigned char *imgOut = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());

  img = image.data;

  switch (op)
  {
    case 1:
          cout<<"Parallel result basic kernel"<<endl;
          start = clock();
          convolution2DGlobalMemKernelCall(image,img,imgOut,h_Mask,Mask_Width,Row,Col);
          finish = clock();
          elapsedParallel = (((double) (finish - start)) / CLOCKS_PER_SEC );
          cout<< "The Secuential process took: " << elapsedParallel << " seconds to execute "<< endl;
          break;

    case 2:
          cout<<"Parallel result with constant mem"<<endl;
          start = clock();
          convolution2DConstantMemKernelCall(image,img,imgOut,h_Mask,Mask_Width,Row,Col);
          finish = clock();
          elapsedParallelConstant = (((double) (finish - start)) / CLOCKS_PER_SEC );
          cout<< "The Secuential process took: " << elapsedParallelConstant << " seconds to execute "<< endl;
          break;

  }

  Mat gray_image;
  gray_image.create(Row,Col,CV_8UC1);
  gray_image.data = imgOut;
  imwrite("./outputs/1053823121.png",gray_image);
  //Wilson if youŕe gonna use this code change the name of the image for your code

  //free(img);
  //free(imgOut);

  return 0;
}
