//============================================================================
#include<cstdlib>
#include<time.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include<math.h> //Included just to use the Power function

#define BLOCK_SIZE 32
#define TILE_SIZE 32
#define MAX_MASK_WIDTH 10
__constant__ float M[MAX_MASK_WIDTH];

using namespace std;

//====== Function made to print vector =========================================
void printVector(float *A, int length)
{
  for (int i=0; i<length; i++)
  {
    cout<<A[i]<<" | ";
  }
  cout<<endl;
}

//====== Function made to fill the vector with some given value ================
void fillVector(float *A, float value, int length)
{
  for (int i=0; i<length; i++)
  {
    A[i] = value;
  }
}

//====== Serial Convolution ====================================================
void serialConvolution(float *input, float *output, float *mask, int mask_length, int length)
{
  int start = 0;
  float temp = 0.0;
  for (int i = 0; i < length; i++)
  {
    for (int j = 0; j < mask_length; j++)
    {
      start = i - (mask_length / 2);
      if (start + j >= 0 && start + j < length)
        temp += input[start + j] * mask[j];
    }
    output[i] = temp;
    temp = 0.0;
  }
}


//====== Basic convolution kernel ==============================================
__global__ void convolutionBasicKernel(float *N, float *M, float *P,
 int Mask_Width, int Width)
 {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   float Pvalue = 0;
   int N_start_point = i - (Mask_Width/2);
   for (int j = 0; j < Mask_Width; j++)
   {
     if (N_start_point + j >= 0 && N_start_point + j < Width)
     {
       Pvalue += N[N_start_point + j]*M[j];
     }
   }
   P[i] = Pvalue;
}

//====== Convolution kernel using constant memory and caching ==================
__global__ void convolutionKernelConstant(float *N, float *P, int Mask_Width,
 int Width)
 {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   float Pvalue = 0;
   int N_start_point = i - (Mask_Width/2);
   for (int j = 0; j < Mask_Width; j++)
   {
     if (N_start_point + j >= 0 && N_start_point + j < Width)
     {
       Pvalue += N[N_start_point + j]*M[j];
     }
   }
   P[i] = Pvalue;
}

//===== Tiled Convolution kernel using shared memory ===========================
__global__ void convolutionKernelShared(float *N, float *P, int Mask_Width,
 int Width)
 {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   __shared__ float N_ds[TILE_SIZE + MAX_MASK_WIDTH - 1];
   int n = Mask_Width/2;
   int halo_index_left = (blockIdx.x - 1)*blockDim.x + threadIdx.x;
   if (threadIdx.x >= blockDim.x - n)
   {
     N_ds[threadIdx.x - (blockDim.x - n)] =
     (halo_index_left < 0) ? 0 : N[halo_index_left];
   }
   N_ds[n + threadIdx.x] = N[blockIdx.x*blockDim.x + threadIdx.x];
   int halo_index_right = (blockIdx.x + 1)*blockDim.x + threadIdx.x;
   if (threadIdx.x < n)
   {
     N_ds[n + blockDim.x + threadIdx.x] =
     (halo_index_right >= Width) ? 0 : N[halo_index_right];
   }
   __syncthreads();
   float Pvalue = 0;
   for(int j = 0; j < Mask_Width; j++)
   {
     Pvalue += N_ds[threadIdx.x + j]*M[j];
   }
   P[i] = Pvalue;
}

//====== A simplier tiled convolution kernel using shared memory and general cahching
__global__ void convolutionKernelSharedSimplier(float *N, float *P, int Mask_Width,
 int Width)
{
 int i = blockIdx.x*blockDim.x + threadIdx.x;
 __shared__ float N_ds[TILE_SIZE];
 N_ds[threadIdx.x] = N[i];
 __syncthreads();
 int This_tile_start_point = blockIdx.x * blockDim.x;
 int Next_tile_start_point = (blockIdx.x + 1) * blockDim.x;
 int N_start_point = i - (Mask_Width/2);
 float Pvalue = 0;
 for (int j = 0; j < Mask_Width; j ++)
 {
   int N_index = N_start_point + j;
   if (N_index >= 0 && N_index < Width)
   {
     if ((N_index >= This_tile_start_point)
     && (N_index < Next_tile_start_point))
     {
       Pvalue += N_ds[threadIdx.x+j-(Mask_Width/2)]*M[j];
     } else
     {
       Pvalue += N[N_index] * M[j];
     }
   }
 }
 P[i] = Pvalue;
}


//===== Convolution kernel call ================================================
void convolutionCall (float *input, float *output, float *mask, int mask_length, int length)
{
  float *d_input;
  float *d_mask;
  float *d_output;
  float block_size = BLOCK_SIZE;//The compiler doesn't let me cast the variable

  hipMalloc(&d_input, length * sizeof(float));
  hipMalloc(&d_mask, mask_length * sizeof(float));
  hipMalloc(&d_output, length * sizeof(float));

  hipMemcpy (d_input, input, length * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpy (d_mask, mask, mask_length * sizeof (float), hipMemcpyHostToDevice);

  dim3 dimGrid (ceil (length / block_size), 1, 1);
  dim3 dimBlock (block_size, 1, 1);

  convolutionBasicKernel<<<dimGrid, dimBlock>>> (d_input, d_mask, d_output, mask_length, length);
  hipDeviceSynchronize();

  hipMemcpy (output, d_output, length * sizeof (float), hipMemcpyDeviceToHost);
  hipFree (d_input);
  hipFree (d_mask);
  hipFree (d_output);
}

//==============================================================================
void convolutionCallWithTiles (float *input, float *output, float *mask, int mask_length, int length)
{
  float *d_input;
  float *d_output;
  float block_size = BLOCK_SIZE;//The compiler doesn't let me cast the variable

  hipMalloc(&d_input, length * sizeof(float));
  hipMalloc(&d_output, length * sizeof(float));

  hipMemcpy (d_input, input, length * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol (HIP_SYMBOL(M), mask, mask_length * sizeof (float));

  dim3 dimGrid (ceil (length / block_size), 1, 1);
  dim3 dimBlock (block_size, 1, 1);

  convolutionKernelSharedSimplier<<<dimGrid, dimBlock>>> (d_input,d_output, mask_length, length);
  hipDeviceSynchronize();

  hipMemcpy (output, d_output, length * sizeof (float), hipMemcpyDeviceToHost);
  hipFree (d_input);
  hipFree (d_output);
}

//==============================================================================
void convolutionCallConstant (float *input, float *output, float *mask, int mask_length, int length)
{
  float *d_input;
  float *d_output;
  float block_size = BLOCK_SIZE;//The compiler doesn't let me cast the variable

  hipMalloc(&d_input, length * sizeof(float));
  hipMalloc(&d_output, length * sizeof(float));

  hipMemcpy (d_input, input, length * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol (HIP_SYMBOL(M), mask, mask_length * sizeof (float));

  dim3 dimGrid (ceil (length / block_size), 1, 1);
  dim3 dimBlock (block_size, 1, 1);

  convolutionKernelConstant<<<dimGrid, dimBlock>>> (d_input,d_output, mask_length, length);
  hipDeviceSynchronize();

  hipMemcpy (output, d_output, length * sizeof (float), hipMemcpyDeviceToHost);
  hipFree (d_input);
  hipFree (d_output);
}

//==============================================================================
void convolutionCallWithTilesComplex (float *input, float *output, float *mask, int mask_length, int length)
{
  float *d_input;
  float *d_output;
  float block_size = BLOCK_SIZE;//The compiler doesn't let me cast the variable

  hipMalloc(&d_input, length * sizeof(float));
  hipMalloc(&d_output, length * sizeof(float));

  hipMemcpy (d_input, input, length * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol (HIP_SYMBOL(M), mask, mask_length * sizeof (float));

  dim3 dimGrid (ceil (length / block_size), 1, 1);
  dim3 dimBlock (block_size, 1, 1);

  convolutionKernelShared<<<dimGrid, dimBlock>>> (d_input,d_output, mask_length, length);
  hipDeviceSynchronize();

  hipMemcpy (output, d_output, length * sizeof (float), hipMemcpyDeviceToHost);
  hipFree (d_input);
  hipFree (d_output);
}

int main ()
{
  int length = 10;
  int mask_length = 5;
  float *A = (float *) malloc(length * sizeof(float));
  float *mask = (float *) malloc(mask_length * sizeof(float));
  float *Cserial = (float *) malloc(length * sizeof(float));
  float *Cparallel = (float *) malloc(length * sizeof(float));
  float *CparallelWithTiles = (float *) malloc(length * sizeof(float));
  float *CparallelConstant = (float *) malloc (length * sizeof(float));
  float *CparallelWithTilesComplex = (float *) malloc(length * sizeof(float));

  fillVector(A,1,length);
  fillVector(mask,2,mask_length);
  fillVector(Cserial,0,length);
  fillVector(Cparallel,0,length);

  serialConvolution(A,Cserial,mask,mask_length,length);
	cout<<"Serial result"<<endl;
  printVector(Cserial,length);

  convolutionCall(A,Cparallel,mask,mask_length,length);
  cout<<"Parallel result"<<endl;
  printVector(Cparallel,length);

  convolutionCallConstant(A,CparallelConstant,mask,mask_length,length);
  cout<<"Parallel with constant memory"<<endl;
  printVector(CparallelConstant,length);

  convolutionCallWithTiles(A,CparallelWithTiles,mask,mask_length,length);
  cout<<"Parallel with shared memory result"<<endl;
  printVector(CparallelWithTiles,length);

  convolutionCallWithTilesComplex(A,CparallelWithTilesComplex,mask,mask_length,length);
  cout<<"Parallel with shared memory result"<<endl;
  printVector(CparallelWithTiles,length);

  free(A);
  free(mask);
  free(Cserial);
  free(Cparallel);
  free(CparallelWithTiles);
  free(CparallelConstant);
  free(CparallelWithTilesComplex);
}
